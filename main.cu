#include <cstdio>
#include <hip/hip_runtime.h>

#define GUARD_CU(a) if (a != hipSuccess) printf("[%s:%d] err(%d) %s\n", __FILE__, __LINE__, a, hipGetErrorString(a));

__global__
void cudaMallocRuntimeTest(int** tab){
    int thid = (blockDim.x * blockIdx.x) + threadIdx.x;
    tab[thid] = (int*)malloc(sizeof(int));
}

__global__
void cudaWriteRuntimeTest(int** tab){
    int thid = (blockDim.x * blockIdx.x) + threadIdx.x;
    tab[thid][0] = 12345;
}

__global__
void cudaReadRuntimeTest(int** tab){
    int thid = (blockDim.x * blockIdx.x) + threadIdx.x;
    printf("%d\n", tab[thid][0]);
}

__global__
void cudaFreeRuntimeTest(int** tab){
    int thid = (blockDim.x * blockIdx.x) + threadIdx.x;
    
    free(tab[thid]);
}

int main(int argn, char* arg[]){
    GUARD_CU(hipDeviceSynchronize());
    GUARD_CU(hipPeekAtLastError());

    int size = 1;

    int** tab = NULL; GUARD_CU(hipMalloc((void**)&tab, sizeof(int*)*size));
    
    cudaMallocRuntimeTest<<<1, size>>>(tab);
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU((hipError_t)hipCtxSynchronize());
    
    cudaWriteRuntimeTest<<<1, size>>>(tab);
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU((hipError_t)hipCtxSynchronize());
    
    cudaReadRuntimeTest<<<1, size>>>(tab);
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU((hipError_t)hipCtxSynchronize());
    
    cudaFreeRuntimeTest<<<1, size>>>(tab);
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU((hipError_t)hipCtxSynchronize());

    GUARD_CU(hipFree(tab));
    return 0;
}
