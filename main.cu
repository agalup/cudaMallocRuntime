#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>

#define GUARD_CU(a) if (a != hipSuccess) printf("[%s:%d] err(%d) %s\n", __FILE__, __LINE__, a, hipGetErrorString(a));

__global__
void cudaMallocRuntimeTest(int** tab){
    int thid = (blockDim.x * blockIdx.x) + threadIdx.x;
    tab[thid] = (int*)malloc(sizeof(int));
}

__global__
void cudaWriteRuntimeTest(int** tab){
    int thid = (blockDim.x * blockIdx.x) + threadIdx.x;
    tab[thid][0] = 12345;
}

__global__
void cudaReadRuntimeTest(int** tab){
    int thid = (blockDim.x * blockIdx.x) + threadIdx.x;
    printf("%d\n", tab[thid][0]);
}

__global__
void cudaFreeRuntimeTest(int** tab){
    int thid = (blockDim.x * blockIdx.x) + threadIdx.x;
    
    free(tab[thid]);
}

int main(int argn, char* arg[]){
    GUARD_CU(hipDeviceSynchronize());
    GUARD_CU(hipPeekAtLastError());

    unsigned int mm_grid_size = 1;
    unsigned int app_grid_size = 1;
    int size = 1;
    int device = 0;

    CUexecAffinityParam_v1 mm_param{CUexecAffinityType::CU_EXEC_AFFINITY_TYPE_SM_COUNT, mm_grid_size};
    CUexecAffinityParam_v1 app_param{CUexecAffinityType::CU_EXEC_AFFINITY_TYPE_SM_COUNT, app_grid_size};
    auto affinity_flags = CUctx_flags::hipDeviceScheduleAuto;

    hipCtx_t mm_ctx, app_ctx;
    GUARD_CU((hipError_t)cuCtxCreate_v3(&mm_ctx, &mm_param, 1, affinity_flags, device));
    GUARD_CU((hipError_t)hipCtxPopCurrent(&mm_ctx));

    GUARD_CU((hipError_t)cuCtxCreate_v3(&app_ctx, &app_param, 1, affinity_flags, device));
    GUARD_CU((hipError_t)hipCtxPopCurrent(&app_ctx));

    int** tab = NULL; GUARD_CU(hipMalloc((void**)&tab, sizeof(int*)*size));
    
    GUARD_CU((hipError_t)hipCtxPushCurrent(mm_ctx));
    cudaMallocRuntimeTest<<<1, size>>>(tab);
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU((hipError_t)hipCtxSynchronize());
    GUARD_CU((hipError_t)hipCtxPopCurrent(&mm_ctx));
    
    GUARD_CU((hipError_t)hipCtxPushCurrent(mm_ctx));
    cudaWriteRuntimeTest<<<1, size>>>(tab);
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU((hipError_t)hipCtxSynchronize());
    GUARD_CU((hipError_t)hipCtxPopCurrent(&mm_ctx));
    
    GUARD_CU((hipError_t)hipCtxPushCurrent(mm_ctx));
    cudaReadRuntimeTest<<<1, size>>>(tab);
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU((hipError_t)hipCtxSynchronize());
    GUARD_CU((hipError_t)hipCtxPopCurrent(&mm_ctx));
    
    GUARD_CU((hipError_t)hipCtxPushCurrent(mm_ctx));
    cudaFreeRuntimeTest<<<1, size>>>(tab);
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU((hipError_t)hipCtxSynchronize());
    GUARD_CU((hipError_t)hipCtxPopCurrent(&mm_ctx));

    GUARD_CU((hipError_t)hipCtxDestroy(mm_ctx));
    GUARD_CU((hipError_t)hipCtxDestroy(app_ctx));

    GUARD_CU(hipFree(tab));

    return 0;
}
